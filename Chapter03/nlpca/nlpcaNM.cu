#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

// define USE_HOST for a host-based version
// define USE_DBL for double-precision
using namespace std;

#include "nelmin.h"
// Define the sigmoidal function
__device__ __host__
inline float G(float x) { return( tanhf(x) ) ;} 
__device__ __host__
inline double G(double x) { return( tanh(x) );} 
// This is a convenience class to hold all the examples and 
// architecture information. Most is boilerplate. CalcError
// is where all the work happens.
template<typename Real>
class ObjFunc {
 private:
  double objFuncCallTime;
  unsigned int objFuncCallCount;
 protected:
  int nExamples;
#ifdef USE_HOST
  thrust::host_vector<Real> h_data;
#else
  thrust::device_vector<Real> d_data;
#endif
  thrust::device_vector<Real> d_param;

 public:
  // The CalcError functor goes here
#include "CalcError.h"
  
  // Boilerplate constructor and helper classes
  ObjFunc() { nExamples = 0; objFuncCallCount=0; objFuncCallTime=0.;}

  double aveObjFuncWallTime() { return(objFuncCallTime/objFuncCallCount); }
  double totalObjFuncWallTime() { return(objFuncCallTime); }
  int get_nExamples() {return(nExamples);}

  void setExamples(thrust::host_vector<Real>& _h_data) {
#ifdef USE_HOST
    h_data = _h_data;
#else
    d_data = _h_data;
#endif
    nExamples = _h_data.size()/exLen;
    d_param = thrust::device_vector<Real>(nParam);
  }

#ifdef USE_HOST
  Real objFunc(Real *p) {
    if(nExamples == 0)  { cerr << "data not set" << endl; exit(1); }

    double startTime=omp_get_wtime();

    Real sum = 0.;
    CalcError getError(&h_data[0], p, nInput, exLen);

#pragma omp parallel for reduction(+ : sum)
    for(int i=0; i < nExamples; ++i) {
      Real d = getError(i);
      sum += d;
    }

    objFuncCallTime += (omp_get_wtime() - startTime);
    objFuncCallCount++;
    return(sum);
  }
#else
  Real objFunc(Real *p)
  {
    if(nExamples == 0)  { cerr << "data not set " <<  endl; exit(1); }

    double startTime=omp_get_wtime();

    thrust::copy(p, p+nParam, d_param.begin());

    CalcError getError(thrust::raw_pointer_cast(&d_data[0]),
		       thrust::raw_pointer_cast(&d_param[0]),
		       nInput, exLen);
    Real sum = thrust::transform_reduce(
			thrust::counting_iterator<unsigned int>(0),
			thrust::counting_iterator<unsigned int>(nExamples),
			getError,
			(Real) 0.,
			thrust::plus<Real>());
    objFuncCallTime += (omp_get_wtime() - startTime);
    objFuncCallCount++;
    return(sum);
  }
#endif
};
// Wrapper so the objective function can be called 
// as a pointer to function for C-style libraries.
// Note: polymorphism allows easy use of 
// either float or double types.
void* objFunc_object=NULL;
float func(float* param)
{
  if(objFunc_object)
    return ((ObjFunc<float>*) objFunc_object)->objFunc(param);
  return(0.);
}
double func(double* param)
{
  if(objFunc_object) 
    return ((ObjFunc<double>*) objFunc_object)->objFunc(param);
  return(0.);
}

// get a uniform random number between -1 and 1 
inline float f_rand() {
  return 2.*(rand()/((float)RAND_MAX)) -1.;
}

template <typename Real>
void genData(thrust::host_vector<Real> &h_data, int nVec, Real xVar)
{
  Real xMax = 1.1; Real xMin = -xMax;
  Real xRange = (xMax - xMin);
  for(int i=0; i < nVec; i++) {
    Real t = xRange * f_rand();
    Real z1 = t +  xVar * f_rand();
    Real z2 = t*t*t +  xVar * f_rand();
    h_data.push_back( z1 ); 
    h_data.push_back( z2 );
  }
}

template <typename Real, int nInput>
void nlpcaDoAll( const Real *p, const Real *in, Real *out) 
{
  register int index=0;
  
  register Real h2_0 = p[index++]; // bottleneck neuron
  {
    register Real h1_0 = p[index++];
    register Real h1_1 = p[index++];
    register Real h1_2 = p[index++];
    register Real h1_3 = p[index++];
    for(int i=0; i < nInput; i++) {
      register Real input=in[i];
      h1_0 += input * p[index++]; h1_1 += input * p[index++];
      h1_2 += input * p[index++]; h1_3 += input * p[index++];
    }
    h1_0 = G(h1_0); h1_1 = G(h1_1);
    h1_2 = G(h1_2); h1_3 = G(h1_3);
    
    h2_0 += p[index++] * h1_0; h2_0 += p[index++] * h1_1;
    h2_0 += p[index++] * h1_2; h2_0 += p[index++] * h1_3;
  }
  
  register Real h3_0 = p[index++];
  register Real h3_1 = p[index++];
  register Real h3_2 = p[index++];
  register Real h3_3 = p[index++];
  h3_0 += p[index++] * h2_0; h3_1 += p[index++] * h2_0;
  h3_2 += p[index++] * h2_0; h3_3 += p[index++] * h2_0;
  h3_0 = G(h3_0); h3_1 = G(h3_1);
  h3_2 = G(h3_2); h3_3 = G(h3_3);
  
  for(int i=0; i < nInput; i++) {
    register Real o = p[index++];
    o += h3_0 * p[index++]; o += h3_1 * p[index++];
    o += h3_2 * p[index++]; o += h3_3 * p[index++];
    out[i]=o;
  }
}

template <typename Real>
void testTraining()
{
  ObjFunc<Real> testObj;
  const int nParam = testObj.nParam;
  cout << "nParam " << nParam << endl;

  // generate the test data
  const int nVec=1000 * 1000 * 10;
  thrust::host_vector<Real> h_data;
  genData<Real>(h_data, nVec, 0.1);
  testObj.setExamples(h_data);
  
  // set the Nelder-Mead starting conditions
  int icount, ifault, numres;
  vector<Real> start(nParam);
  vector<Real> step(nParam,1.);
  vector<Real> xmin(nParam);

  srand(0);
  for(int i=0; i < start.size(); i++) start[i] = 0.2 * f_rand();

  Real ynewlo = testObj.objFunc( &start[0] );
  Real reqmin = 1.0E-18;
  int konvge = 10;
  int kcount = 100000;

  objFunc_object = &testObj;
  double optStartTime=omp_get_wtime();
  nelmin<Real> (func, nParam, &start[0], &xmin[0], &ynewlo, reqmin, &step[0],
		  konvge, kcount, &icount, &numres, &ifault );
  double optTime=omp_get_wtime()-optStartTime;

  cout << endl <<"  Return code IFAULT = " << ifault << endl << endl;
  cout << "  Estimate of minimizing value X*:" << endl << endl;
  cout << "  F(X*) = " << ynewlo << endl;
  cout << "  Number of iterations = " << icount << endl;
  cout << "  Number of restarts =   " << numres << endl << endl;

  cout << "Average wall time for ObjFunc " 
       << testObj.aveObjFuncWallTime() << endl;
  cout << "Total wall time in optimization method " << optTime << endl;
  cout << "Percent time in objective function " << 
    (100.*(testObj.totalObjFuncWallTime()/optTime)) << endl;
  
  cout << " -- Generate scatter plot  -- " << endl;
  cout << "original known pred" << endl;
  thrust::host_vector<Real> h_test;
  thrust::host_vector<Real> h_in(testObj.nInput);
  thrust::host_vector<Real> h_out(testObj.nInput);

  int index=0, nTest=100;
  genData<Real>(h_test, nTest, 0.0); // note: no variance for the test
  for(int i=0; i< nTest; i++) {
    h_in[0] = h_test[index++];
    h_in[1] = h_test[index++];
    
    nlpcaDoAll<Real,2>(&xmin[0],&h_in[0],&h_out[0]);
    cout << h_data[testObj.nInput*i] << "," << h_data[testObj.nInput*i+1] << " "
	 << h_in[0] << "," << h_in[1] << " "
	 << h_out[0] << "," << h_out[1] << endl;
  }
}

int main ( )
{
#ifdef USE_DBL
  testTraining<double> ( );
#else
  testTraining<float> ( );
#endif
  return 0;
}
