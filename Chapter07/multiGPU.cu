#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <thrust/reduce.h>

__global__ void fillKernel(int *a, int n, int offset)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;
  if (tid < n)
      a[tid] = offset+tid;
}

int main(int argc, char* argv[])
{
  int nGPU;
  int n = 1000000;
  int size = n * sizeof(int);
  
  hipGetDeviceCount(&nGPU);
  
  int *d_A[nGPU];
  for(int i=0; i < nGPU; i++) {
    hipSetDevice(i);
    if(hipSetDeviceFlags(hipDeviceScheduleYield)) {
      fprintf(stderr,"hipSetDeviceFlags failed!\n"); exit(1);
    }
    hipMalloc(&d_A[i],size);
  }
  
  int *h_A;
  hipHostAlloc(&h_A, nGPU*size, hipHostMallocMapped);
  
  for(int i=0; i < nGPU; i++) {
    int nThreadsPerBlock= 512;
    int nBlocks= n/nThreadsPerBlock + ((n%nThreadsPerBlock)?1:0);
    hipSetDevice(i);
    fillKernel<<<nBlocks, nThreadsPerBlock>>>(d_A[i], n, i*n);
  }
  double sTime = omp_get_wtime();
  for(int i=0; i < nGPU; i++) {
    hipSetDevice(i);
    hipMemcpyAsync(&h_A[i*n], d_A[i], size, hipMemcpyDeviceToHost);
  }
  hipDeviceSynchronize();
  double eTime = omp_get_wtime();
  printf("time %f\n", eTime-sTime);

  for(int i=0; i < nGPU*n; i++)
    if(h_A[i] != i) {
      printf("Error h_A[%d] = %d\n",i,h_A[i]); exit(1);
    }
  printf("Success!\n");

  hipHostFree(h_A);
  for(int i=0; i < nGPU; i++) {
    hipSetDevice(i);
    hipFree(d_A[i]);
  }
  return(0);
}
